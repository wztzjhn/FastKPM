#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>

namespace fkpm {

    template<typename T>
    __device__ __forceinline__ T ldg(const T* ptr) {
#if __CUDA_ARCH__ >= 350
        return __ldg(ptr);
#else
        return *ptr;
#endif
    }

    template <typename T> T zero();
    template<> __device__ __forceinline__ float           zero()                         { return 0; }
    template<> __device__ __forceinline__ double          zero()                         { return 0; }
    template<> __device__ __forceinline__ hipFloatComplex  zero()                         { return make_hipFloatComplex(0, 0); }
    template<> __device__ __forceinline__ hipDoubleComplex zero()                         { return make_hipDoubleComplex(0, 0); }

    __device__ __forceinline__ float           add(float a, float b)                     { return a+b; }
    __device__ __forceinline__ double          add(double a, double b)                   { return a+b; }
    __device__ __forceinline__ hipFloatComplex  add(hipFloatComplex a,  hipFloatComplex b)  { return hipCaddf(a, b); }
    __device__ __forceinline__ hipDoubleComplex add(hipDoubleComplex a, hipDoubleComplex b) { return hipCadd(a, b); }

    __device__ __forceinline__ float           conj(float a)                             { return a; }
    __device__ __forceinline__ double          conj(double a)                            { return a; }
    __device__ __forceinline__ hipFloatComplex  conj(hipFloatComplex a)                    { return hipConjf(a); }
    __device__ __forceinline__ hipDoubleComplex conj(hipDoubleComplex a)                   { return hipConj(a); }

    __device__ __forceinline__ float           mul(float a, float b)                     { return a*b; }
    __device__ __forceinline__ double          mul(double a, double b)                   { return a*b; }
    __device__ __forceinline__ hipFloatComplex  mul(hipFloatComplex a,  hipFloatComplex b)  { return hipCmulf(a, b); }
    __device__ __forceinline__ hipDoubleComplex mul(hipDoubleComplex a, hipDoubleComplex b) { return hipCmul(a, b); }
    __device__ __forceinline__ hipFloatComplex  mul(float a,  hipFloatComplex b)           { return make_hipFloatComplex(a*b.x, a*b.y); }
    __device__ __forceinline__ hipDoubleComplex mul(double a, hipDoubleComplex b)          { return make_hipDoubleComplex(a*b.x, a*b.y); }


    template <typename T, typename T_re>
    __global__ void outer_product_kernel(int b_rows, int b_len, int n_cols, T_re alpha, const T *A, const T *B,
                                         int n_blocks, const int *D_row_idx, const int *D_col_idx, T *D_val) {
        // idx realizes each index into D_val
        for (int idx = blockIdx.x*blockDim.x + threadIdx.x;
             idx < b_len*b_len*n_blocks;
             idx += gridDim.x*blockDim.x) {

            int k = idx / (b_len*b_len);
            int i = ldg(D_row_idx + k);
            int j = ldg(D_col_idx + k);
            int bj = (idx / b_len) % b_len;
            int bi = idx % b_len;

            T acc = zero<T>();
            for (int l = 0; l < n_cols; l++) {
                T a = ldg(A + n_cols*(b_len*i + bi) + l);
                T b = ldg(B + n_cols*(b_len*j + bj) + l);
                acc = add(acc, mul(a, conj(b)));
            }
            D_val[idx] = add(D_val[idx], mul(alpha, acc));
        }
    }

    template <typename T, typename T_re>
    void outer_product(int b_rows, int b_len, int n_cols, T_re alpha, const T *A, const T *B,
                       int n_blocks, const int *D_row_idx, const int *D_col_idx, T *D_val) {
        int block_size = 64;
        int grid_size = min(max(b_len*b_len*n_blocks / block_size, 1), 256);
        outer_product_kernel<<<grid_size, block_size>>>(b_rows, b_len, n_cols, alpha, A, B, n_blocks, D_row_idx, D_col_idx, D_val);
    }

    template void outer_product(int b_rows, int b_len, int n_cols, float alpha, const float *A, const float *B,
                                int n_blocks, const int *D_row_idx, const int *D_col_idx, float *D_val);

    template void outer_product(int b_rows, int b_len, int n_cols, double alpha, const double *A, const double *B,
                                int n_blocks, const int *D_row_idx, const int *D_col_idx, double *D_val);

    template void outer_product<hipFloatComplex,float>(int b_rows, int b_len, int n_cols, float alpha, const hipFloatComplex *A, const hipFloatComplex *B,
                                int n_blocks, const int *D_row_idx, const int *D_col_idx, hipFloatComplex *D_val);

    template void outer_product(int b_rows, int b_len, int n_cols, double alpha, const hipDoubleComplex *A, const hipDoubleComplex *B,
                                int n_blocks, const int *D_row_idx, const int *D_col_idx, hipDoubleComplex *D_val);
}
